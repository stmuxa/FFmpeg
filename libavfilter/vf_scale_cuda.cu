#include "hip/hip_runtime.h"
/*
 * This file is part of FFmpeg.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "cuda/vector_helpers.cuh"

// FFmpeg passes pitch in bytes, CUDA uses potentially larger types
#define FIXED_PITCH \
    (dst_pitch/sizeof(*dst_0))

template<typename T>
__device__ static inline T Subsample_Bilinear(hipTextureObject_t tex,
                                              int xo, int yo,
                                              int dst_width, int dst_height,
                                              int src_width, int src_height,
                                              int bit_depth, float param)
{
    float hscale = (float)src_width / (float)dst_width;
    float vscale = (float)src_height / (float)dst_height;
    float xi = (xo + 0.5f) * hscale;
    float yi = (yo + 0.5f) * vscale;
    // 3-tap filter weights are {wh,1.0,wh} and {wv,1.0,wv}
    float wh = min(max(0.5f * (hscale - 1.0f), 0.0f), 1.0f);
    float wv = min(max(0.5f * (vscale - 1.0f), 0.0f), 1.0f);
    // Convert weights to two bilinear weights -> {wh,1.0,wh} -> {wh,0.5,0} + {0,0.5,wh}
    float dx = wh / (0.5f + wh);
    float dy = wv / (0.5f + wv);

    intT r;
    vec_set_scalar(r, 2);
    r += tex2D<T>(tex, xi - dx, yi - dy);
    r += tex2D<T>(tex, xi + dx, yi - dy);
    r += tex2D<T>(tex, xi - dx, yi + dy);
    r += tex2D<T>(tex, xi + dx, yi + dy);

    T res;
    vec_set(res, r >> 2);

    return res;
}

extern "C" {

__constant__ uchar font[11][8][8] = {
    { // 0
        0,0,0,0,0,0,0,0,
        0,0,0,1,1,0,0,0,
        0,0,1,0,0,1,0,0,
        0,0,1,0,0,1,0,0,
        0,0,1,0,0,1,0,0,
        0,0,1,0,0,1,0,0,
        0,0,0,1,1,0,0,0,
        0,0,0,0,0,0,0,0,
    },
    { // 1
        0,0,0,0,0,0,0,0,
        0,0,0,0,1,0,0,0,
        0,0,0,1,1,0,0,0,
        0,0,0,0,1,0,0,0,
        0,0,0,0,1,0,0,0,
        0,0,0,0,1,0,0,0,
        0,0,0,1,1,1,0,0,
        0,0,0,0,0,0,0,0,
    },
    { // 2
        0,0,0,0,0,0,0,0,
        0,0,1,1,1,0,0,0,
        0,0,0,0,0,1,0,0,
        0,0,0,1,1,0,0,0,
        0,0,1,0,0,0,0,0,
        0,0,1,0,0,0,0,0,
        0,0,0,1,1,1,0,0,
        0,0,0,0,0,0,0,0,
    },
    { // 3
        0,0,0,0,0,0,0,0,
        0,0,1,1,1,0,0,0,
        0,0,0,0,0,1,0,0,
        0,0,0,1,1,0,0,0,
        0,0,0,0,0,1,0,0,
        0,0,0,0,0,1,0,0,
        0,0,1,1,1,0,0,0,
        0,0,0,0,0,0,0,0,
    },
    { // 4
        0,0,0,0,0,0,0,0,
        0,0,0,0,1,0,0,0,
        0,0,0,1,1,0,0,0,
        0,0,1,0,1,0,0,0,
        0,1,1,1,1,1,0,0,
        0,0,0,0,1,0,0,0,
        0,0,0,0,1,0,0,0,
        0,0,0,0,0,0,0,0,
    },
    { // 5
        0,0,0,0,0,0,0,0,
        0,0,0,1,1,1,0,0,
        0,0,1,0,0,0,0,0,
        0,0,0,1,1,0,0,0,
        0,0,0,0,0,1,0,0,
        0,0,0,0,0,1,0,0,
        0,0,1,1,1,0,0,0,
        0,0,0,0,0,0,0,0,
    },
    { // 6
        0,0,0,0,0,0,0,0,
        0,0,0,1,1,0,0,0,
        0,0,1,0,0,0,0,0,
        0,0,1,1,1,0,0,0,
        0,0,1,0,0,1,0,0,
        0,0,1,0,0,1,0,0,
        0,0,0,1,1,0,0,0,
        0,0,0,0,0,0,0,0,
    },
    { // 7
        0,0,0,0,0,0,0,0,
        0,0,1,1,1,1,0,0,
        0,0,0,0,1,0,0,0,
        0,0,0,1,0,0,0,0,
        0,0,0,1,0,0,0,0,
        0,0,0,1,0,0,0,0,
        0,0,0,1,0,0,0,0,
        0,0,0,0,0,0,0,0,
    },
    { // 8
        0,0,0,0,0,0,0,0,
        0,0,0,1,1,0,0,0,
        0,0,1,0,0,1,0,0,
        0,0,0,1,1,0,0,0,
        0,0,1,0,0,1,0,0,
        0,0,1,0,0,1,0,0,
        0,0,0,1,1,0,0,0,
        0,0,0,0,0,0,0,0,
    },
    { // 9
        0,0,0,0,0,0,0,0,
        0,0,0,1,1,0,0,0,
        0,0,1,0,0,1,0,0,
        0,0,1,0,0,1,0,0,
        0,0,0,1,1,1,0,0,
        0,0,0,0,0,1,0,0,
        0,0,0,1,1,0,0,0,
        0,0,0,0,0,0,0,0,
    },
    { // Slash
        0,0,0,0,0,0,0,0,
        0,0,0,0,0,1,0,0,
        0,0,0,0,0,1,0,0,
        0,0,0,0,1,0,0,0,
        0,0,0,0,1,0,0,0,
        0,0,0,1,0,0,0,0,
        0,0,0,1,0,0,0,0,
        0,0,0,0,0,0,0,0,
    },
};

__global__ void Subsample_Bilinear_nv12_nv12(
        hipTextureObject_t src_tex_0, hipTextureObject_t src_tex_1,
        hipTextureObject_t src_tex_2, hipTextureObject_t src_tex_3,
        uchar *dst_0, uchar *dst_1, uchar *dst_2, uchar *dst_3,
        int dst_width, int dst_height, int dst_pitch,
        int src_width, int src_height, float param,
        char *frame_rate_num, int frame_rate_num_digits,
        char *frame_rate_den, int frame_rate_den_digits)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;
    if (yo >= dst_height || xo >= dst_width) return;

    uchar y = Subsample_Bilinear<uchar>(src_tex_0, xo, yo, dst_width, dst_height, src_width, src_height, 8, 0.0f);

    // Draw FPS
    int font_scale_log2 = 2;
    int frame_rate_msg_size = frame_rate_num_digits + 1 + frame_rate_den_digits;
    int font_size = 8 << font_scale_log2;

    if (xo < frame_rate_msg_size * font_size && yo < font_size)
    {
        int symbol_pos = xo / font_size;
        char symbol = 10; // Slash
        if (symbol_pos < frame_rate_num_digits)
        {
            symbol = frame_rate_num[symbol_pos] - '0';
        }
        else if (symbol_pos > frame_rate_num_digits)
        {
            symbol = frame_rate_den[symbol_pos - 1 - frame_rate_num_digits] - '0';
        }

        int fontx = (xo>>font_scale_log2) & 0x7;
        int fonty = (yo>>font_scale_log2) & 0x7;

        y = font[symbol][fonty][fontx] * 255;
    }

    dst_0[yo*FIXED_PITCH+xo] = y;
}

__global__ void Subsample_Bilinear_nv12_nv12_uv(
        hipTextureObject_t src_tex_0, hipTextureObject_t src_tex_1,
        hipTextureObject_t src_tex_2, hipTextureObject_t src_tex_3,
        uchar2 *dst_0, uchar2 *dst_1, uchar2 *dst_2, uchar2 *dst_3,
        int dst_width, int dst_height, int dst_pitch,
        int src_width, int src_height, float param,
        char *frame_rate_num, int frame_rate_num_digits,
        char *frame_rate_den, int frame_rate_den_digits)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;
    if (yo >= dst_height || xo >= dst_width) return;

    uchar2 uv = Subsample_Bilinear<uchar2>(src_tex_1, xo, yo, dst_width, dst_height, src_width, src_height, 8, 0.0f);

    dst_1[yo*FIXED_PITCH+xo].x = uv.x;
    dst_1[yo*FIXED_PITCH+xo].y = uv.y;
}
}
